
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <algorithm>
#define DEBUG_ON
#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true);


/*******************************************************************
  * Matrix configuration
  ******************************************************************/

#define DTYPE float
const int M = 8;
const int N = 8;
const int K = 8;
const float sparsity = 0.2;
const int warp_size = 5;

/*******************************************************************
  * Kernel code
  ******************************************************************/
template <typename T>
__global__ void get_num_nonzero_by_row(T* A, int* row_ptr, int M, int K) {


    int y = blockIdx.y * blockDim.y + threadIdx.y; // row
    int x = threadIdx.x; // warp

    int K_under = (K/blockDim.x)*blockDim.x;
    int sum = 0;
    unsigned int bits;

    for (int i=0; i<K_under; i+=blockDim.x) {
        // warp communication
        bits = __ballot_sync(0xFFFFFFFF, A[y*K+(x+i)] != 0);
        if (x == 0){
            sum += __popc(bits);
        }
    }

    if (x == 0) {
        for (int i=K_under; i<K; i++) {
            if (A[y*K+i] != 0)
                sum++;
        }
        row_ptr[y+1] = sum;
    }
    
}


template <typename T, int WARP_SIZE>
__global__ void store_nonzero_by_row(T* A, int* row_ptr, int* col, T* value, int M, int K) {


    int y = blockIdx.y * blockDim.y + threadIdx.y; // row
    int x = threadIdx.x; // warp

    __shared__ float svalue[WARP_SIZE];

    int K_under = (K/blockDim.x)*blockDim.x;
    int num = 0;

    for (int i=0; i<K_under; i+=blockDim.x) {
        // warp communication
        svalue[x] = A[y*K+(x+i)];
        __syncthreads();
        if (x == 0){
            
            #pragma unroll
            for (int w=0; w<WARP_SIZE; w++) {
                if (svalue[w] != 0) {
                    printf("[%d]%d %d : %f\n",row_ptr[y]+num, y, (w+i), svalue[w]);
                    col[row_ptr[y]+num] = (w+i);
                    value[row_ptr[y]+num] = svalue[w];
                    num++;
                }
                    
            }

        }
    }

    if (x == 0) {
        for (int w=K_under; w<K; w++) {
            if (A[y*K+w] != 0){
                printf("[%d]%d %d : %f\n", row_ptr[y]+num, y, w, A[y*K+w]);
                col[row_ptr[y]+num] = w;
                value[row_ptr[y]+num] = A[y*K+w];
                num++;   
            }
        }
    }


}

/*******************************************************************
  * Host code
  ******************************************************************/

void print_matrix(const std::vector<DTYPE>& A, int ROW);
void print_vector(const std::vector<int>& A);
DTYPE get_random_number() {return std::rand()%10-5;}
void make_sparse_matrix(std::vector<DTYPE>& A);
void conversion_CSR(DTYPE* d_A, void** d_row_ptr_p, void** d_col_p, void** d_value_p);


/*******************************************************************
  * Main
  ******************************************************************/

int main(void) {

    std::cout << "" << std::endl;
    std::cout << "==========================================================" << std::endl;
    std::cout << "Sparse Matrix Multipliation Example" << std::endl;
    std::cout << "  -- Mutiplication of a sparse matrix and a dense matrix" << std::endl;
    std::cout << "  -- C["<<M<<","<<N<<"] = A["<<M<<","<<K<<"] * B["<<K<<","<<N<<"]" << std::endl;
    std::cout << "  -- Sparsity of matrix : " << sparsity << std::endl;
    std::cout << "==========================================================" << std::endl;
    std::cout << "" << std::endl;

    /* Data initialization */
    std::vector<DTYPE> A(M*K);
    make_sparse_matrix(A);
    std::vector<DTYPE> B(K*N);
    std::generate(B.begin(), B.end(), get_random_number);
    std::vector<DTYPE> C(M*N);

    /* Alloc GPU memory */
    DTYPE *d_A, *d_B, *d_C;
    cudaErrChk( hipMalloc((void**)&d_A, sizeof(DTYPE)*M*K) );
    cudaErrChk( hipMalloc((void**)&d_B, sizeof(DTYPE)*K*N) );
    cudaErrChk( hipMalloc((void**)&d_C, sizeof(DTYPE)*M*N) );
    
    /* Memcpy from host to device */
    cudaErrChk( hipMemcpy(d_A, A.data(), sizeof(DTYPE)*M*K, hipMemcpyHostToDevice) );
    cudaErrChk( hipMemcpy(d_B, B.data(), sizeof(DTYPE)*K*N, hipMemcpyHostToDevice) );
    cudaErrChk( hipDeviceSynchronize() );
    cudaErrChk( hipGetLastError() );


    /*****************************
     * Conversion 
     *****************************/
    DTYPE *d_row_ptr, *d_col, *d_value;
    conversion_CSR(d_A, (void**)&d_row_ptr, (void**)&d_col, (void**)&d_value);


    /*****************************
     * Kernel code
     *****************************/




    /* Finalize */
    cudaErrChk( hipFree(d_A) );
    cudaErrChk( hipFree(d_B) );
    cudaErrChk( hipFree(d_C) );
    cudaErrChk( hipFree(d_row_ptr) );
    cudaErrChk( hipFree(d_col) );
    cudaErrChk( hipFree(d_value) );
    return 0;
}


/*******************************************************************
  * Host code
  ******************************************************************/
DTYPE get_nonzero() {
    while (true) {
        DTYPE value = std::rand()%10-5;
        if (value != 0)
            return value;
    }
}

void make_sparse_matrix(std::vector<DTYPE>& A) {

    int total_nonzero = sparsity * M * K;

    std::vector<int> idx(M*K);
    for (int i=0; i<M*K; i++) idx[i] = i;

    // Select matrix indices for nonzero value
    while (idx.size() != total_nonzero) {
        idx.erase(idx.begin() + std::rand()%idx.size());
    }

    // Fill nonzero value into selected indices
    for (int i=0; i<total_nonzero; i++) {
        A[idx[i]] = get_nonzero();
    }

    print_matrix(A, M);
}

void conversion_CSR(DTYPE* d_A, void** d_row_ptr_p, void** d_col_p, void** d_value_p) {

    printf("CSR conversion launched...\n");

    
    float msec_total = 0.0f;
    hipEvent_t start, stop;
    cudaErrChk( hipEventCreate(&start) );
    cudaErrChk( hipEventCreate(&stop) );
    cudaErrChk( hipEventRecord(start, NULL) );

    /*** Start of conversion ***/
    std::vector<int> row_ptr(M+1);
    cudaErrChk( hipMalloc(d_row_ptr_p, sizeof(int)*(M+1)) );
    const dim3 dim_threads(warp_size, 1);
    const dim3 dim_blocks(1, M);
    
    // Count the number of non-zero values by rows
    get_num_nonzero_by_row<DTYPE><<<dim_blocks, dim_threads>>>(d_A, (int*)(*d_row_ptr_p), M, K);
    cudaErrChk( hipMemcpy(row_ptr.data(),(*d_row_ptr_p), sizeof(int)*(M+1), hipMemcpyDeviceToHost) );
    cudaErrChk( hipDeviceSynchronize() );
    cudaErrChk( hipGetLastError() );
    print_vector(row_ptr);

    // Exclusive scan
    row_ptr[0] = 0;
    for (int i=1; i<(M+1); i++)
        row_ptr[i] += row_ptr[i-1];
    print_vector(row_ptr);
    cudaErrChk( hipMemcpy((*d_row_ptr_p), row_ptr.data(), sizeof(int)*(M+1), hipMemcpyHostToDevice) );
    cudaErrChk( hipDeviceSynchronize() );
    cudaErrChk( hipGetLastError() );

    // Store non-zero values
    cudaErrChk( hipMalloc(d_col_p, sizeof(int)*row_ptr[M]) );
    cudaErrChk( hipMalloc(d_value_p, sizeof(DTYPE)*row_ptr[M]) );
    store_nonzero_by_row<DTYPE, warp_size><<<dim_blocks, dim_threads>>>(d_A, (int*)(*d_row_ptr_p), (int*)(*d_col_p), (DTYPE*)(*d_value_p), M, K);



    /*** End of conversion ***/
    cudaErrChk( hipEventRecord(stop, NULL) );
    cudaErrChk( hipEventSynchronize(stop) );
    cudaErrChk( hipEventElapsedTime(&msec_total, start, stop) );
    printf(" -- Elapsed time: %.3f s\n", msec_total*1e-3);

}




/*******************************************************************
  * Debug code
  ******************************************************************/

inline void cudaAssert(hipError_t code, const char *file, int line, bool abort) {
    if (code != hipSuccess) {
       fprintf(stderr,"CUDA assert: %s %s %d\n", hipGetErrorString(code), file, line);
       if (abort) exit(code);
    }
}

void print_matrix(const std::vector<DTYPE>& A, int ROW) {

    int COL = A.size() / ROW;

    for (int row=0; row<ROW; row++) {
        for (int col=0; col<COL; col++) {
            std::cout << A[row*COL+col] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

void print_vector(const std::vector<int>& A) {

    for (int i=0; i<A.size(); i++)
        std::cout << A[i] << " ";

    std::cout << std::endl << std::endl;
}