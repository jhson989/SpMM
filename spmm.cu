/********************************************************************************
  * Main
  * Sparse Matrix Multipliation Example
  * Author : Janghyun Son
  * Email : jhson989@gmail.com
  *******************************************************************************/


#include <algorithm>
#include <time.h>

#include "include/config.cuh" // Program configuration 
#include "include/debug.cuh"  // Debug code
#include "include/data.cuh" // Sparse matrix generator
#include "include/convert.cuh" // CSR convertor
#include "include/matmul_sparse.cuh" // SpMM implementation



int main(void) {

    /*******************************************************************
     * Log 
     *******************************************************************/

    srand(time(NULL));
    std::cout << "" << std::endl;
    std::cout << "==========================================================" << std::endl;
    std::cout << "Sparse Matrix Multipliation Example" << std::endl;
    std::cout << "  -- Mutiplication of a sparse matrix and a dense matrix" << std::endl;
    std::cout << "  -- C["<<M<<","<<N<<"] = A["<<M<<","<<K<<"] * B["<<K<<","<<N<<"]" << std::endl;
    std::cout << "  -- Sparsity of matrix : " << SPARSITY << std::endl;
    std::cout << "==========================================================" << std::endl;
    std::cout << "" << std::endl;



    /*******************************************************************
     * Data initialization
     *******************************************************************/

    /* Host data generation */
    std::vector<DTYPE> A(M*K);
    make_sparse_matrix(A);
    std::vector<DTYPE> B(K*N);
    std::generate(B.begin(), B.end(), get_random_number);
    std::vector<DTYPE> C(M*N, 0);

    /* Alloc GPU memory */
    DTYPE *d_A, *d_B, *d_C;
    cudaErrChk( hipMalloc((void**)&d_A, sizeof(DTYPE)*M*K) );
    cudaErrChk( hipMalloc((void**)&d_B, sizeof(DTYPE)*K*N) );
    cudaErrChk( hipMalloc((void**)&d_C, sizeof(DTYPE)*M*N) );
    
    /* Memcpy from host to device */
    cudaErrChk( hipMemcpy(d_A, A.data(), sizeof(DTYPE)*M*K, hipMemcpyHostToDevice) );
    cudaErrChk( hipMemcpy(d_B, B.data(), sizeof(DTYPE)*K*N, hipMemcpyHostToDevice) );
    cudaErrChk( hipDeviceSynchronize() );
    cudaErrChk( hipGetLastError() );



    /*******************************************************************
     * Conversion 
     *******************************************************************/
    
    /* Device memory for CSR format array : rowPtr, col, value */
    int *d_row_ptr, *d_col; 
    DTYPE *d_value;
    
    /* Run CSR convertor */
    convert_to_CSR(d_A, (void**)&d_row_ptr, (void**)&d_col, (void**)&d_value);



    /*******************************************************************
     * Sparse - Dense Matrix Multiplication
     *******************************************************************/

    /* Run SpMM CPU implementation */
    spmm_cpu(d_row_ptr, d_col, d_value, A, B, C);

    /* Run SpMM GPU implementation - 1 */
    spmm_gpu_1(d_row_ptr, d_col, d_value, d_A, d_B, d_C, A, B, C);

    /*******************************************************************
     * Finalize
     *******************************************************************/

    /* Dealloc memory */
    cudaErrChk( hipFree(d_A) );
    cudaErrChk( hipFree(d_B) );
    cudaErrChk( hipFree(d_C) );
    cudaErrChk( hipFree(d_row_ptr) );
    cudaErrChk( hipFree(d_col) );
    cudaErrChk( hipFree(d_value) );
    return 0;
}
  